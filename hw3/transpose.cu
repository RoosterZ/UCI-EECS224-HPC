#include "hip/hip_runtime.h"
#include <stdlib.h>
#include <stdio.h>

#include "cuda_utils.h"
#include "timer.c"

#define MIN(x,y) ((x < y) ? x : y)
#define MAX(x,y) ((x > y) ? x : y)
#define MIN_THREAD 32
#define MAX_THREAD 32
//#define CASCADING 8
#define SCRATCH_SIZE 4160
#define BLOCK_DIM_Y 8
#define PATCH_DIM 32


typedef float dtype;


unsigned int nextPow2( unsigned int x ) {
	--x;
	x |= x >> 1;
	x |= x >> 2;
	x |= x >> 4;
	x |= x >> 8;
	x |= x >> 16;
	return ++x;
}
  
void getNumBlocksAndThreads(unsigned int dim, int &bx, int &by, int &gx, int &gy){
	//int blockx, blocky, gridx, gridy;
	bx = dim < MAX_THREAD ? MAX(nextPow2(dim), MIN_THREAD) : MAX_THREAD;
	by = BLOCK_DIM_Y;
	//gridx = ceil(dim / float(blockx));
	gx = (dim + bx - 1) / bx;

	//gridy = ceil(dim / float(CASCADING));
	gy = gx;
	//cout << bx << " " << gridx << " " << gridy;
}

// __global__ 
// void matTrans(dtype* AT, dtype* A, int N)  {
// 	/* Fill your code here */
// 	//const unsigned int scratch_dim = blockDim.x;
// 	//__shared__ dtype scratch[scratch_dim][scratch_dim + 1];
// 	__shared__ dtype scratch[32][33];
// 	int x = blockIdx.x * blockDim.x + threadIdx.x;
// 	int y = blockIdx.y * blockDim.x + threadIdx.y;

	
// 	int i;
// 	//int dim = gridDim.x * blockDim.x;
// 	for (i = 0; i < blockDim.x; i += BLOCK_DIM_Y){
// 		scratch[i + threadIdx.y][threadIdx.x] = A[(y+i) * N + x]; 
// 	}

// 	__syncthreads();

// 	x = blockDim.x * blockIdx.y + threadIdx.x;
// 	y = blockIdx.x * blockDim.x + threadIdx.y;
 
// 	for (i = 0; i < blockDim.x; i += BLOCK_DIM_Y){
// 		AT[(y+i) * N + x] = scratch[threadIdx.x][i + threadIdx.y];
// 	}
    

// }

__global__ 
void matTrans(dtype* AT, dtype* A, int N)  {
	/* Fill your code here */
	//const unsigned int scratch_dim = blockDim.x;
	
	//__shared__ dtype scratch[PATCH_DIM][PATCH_DIM+1];
	int x = blockIdx.x * PATCH_DIM + threadIdx.x;
	int y = blockIdx.y * PATCH_DIM + threadIdx.y;

	int i;
	//int dim = gridDim.x * blockDim.x;
	// for (i = 0; i < PATCH_DIM; i += BLOCK_DIM_Y){
	// 	scratch[i + threadIdx.y][threadIdx.x] = A[(y+i) * N + x]; 
	// }
	for (i = 0; i < PATCH_DIM; i += BLOCK_DIM_Y){
		AT[(y+i) * N + x] = A[(y+i) * N + x]; 
	}	

	// __syncthreads();

	// x = PATCH_DIM * blockIdx.y + threadIdx.x;
	// y = blockIdx.x * PATCH_DIM + threadIdx.y;
 
	// for (i = 0; i < PATCH_DIM; i += BLOCK_DIM_Y){
	// 	AT[(y+i) * N + x] = scratch[threadIdx.x][i + threadIdx.y];
	// }
    

}

// __global__ void copy(float *odata, const float *idata)
// {
//   int x = blockIdx.x * TILE_DIM + threadIdx.x;
//   int y = blockIdx.y * TILE_DIM + threadIdx.y;
//   int width = gridDim.x * TILE_DIM;

//   for (int j = 0; j < TILE_DIM; j+= BLOCK_ROWS)
//     odata[(y+j)*width + x] = idata[(y+j)*width + x];
// }


void
parseArg (int argc, char** argv, int* N)
{
	if(argc == 2) {
		*N = atoi (argv[1]);
		assert (*N > 0);
	} else {
		fprintf (stderr, "usage: %s <N>\n", argv[0]);
		exit (EXIT_FAILURE);
	}
}

void
initArr (dtype* in, int N)
{
	int i;

	for(i = 0; i < N; i++) {
		in[i] = (dtype) rand () / RAND_MAX;
	}
}

void
cpuTranspose (dtype* A, dtype* AT, int N)
{
	int i, j;

	for(i = 0; i < N; i++) {
		for(j = 0; j < N; j++) {
			AT[j * N + i] = A[i * N + j];
		}
	}
}

int
cmpArr (dtype* a, dtype* b, int N)
{
	int cnt, i;

	cnt = 0;
	for(i = 0; i < N; i++) {
		if(abs(a[i] - b[i]) > 1e-6) cnt++;
	}

	return cnt;
}

void
gpuTranspose (dtype* A, dtype* AT, int N)
{
	dtype *d_idata, *d_odata;
	CUDA_CHECK_ERROR (hipMalloc (&d_idata, N * N * sizeof (dtype)));
	CUDA_CHECK_ERROR (hipMalloc (&d_odata, N * N * sizeof (dtype)));
	CUDA_CHECK_ERROR (hipMemcpy (d_idata, A, N * N * sizeof (dtype), 
	hipMemcpyHostToDevice));

	int block_x, block_y, grid_x, grid_y;
	getNumBlocksAndThreads(N, block_x, block_y, grid_x, grid_y);
	// dim3 gb(grid_x, grid_y, 1);
	// dim3 tb(block_x, block_y, 1);
	dim3 gb(32, 8, 1);
	dim3 tb(32, 32, 1);
	matTrans <<<gb, tb>>> (d_odata, d_idata, N);

	struct stopwatch_t* timer = NULL;
  	long double t_gpu;
	
  	/* Setup timers */
  	stopwatch_init ();
  	timer = stopwatch_create ();
  
	stopwatch_start (timer);
	  
	matTrans <<<gb, tb>>> (d_odata, d_idata, N);
	/* run your kernel here */

  	hipDeviceSynchronize ();
  	t_gpu = stopwatch_stop (timer);
  	fprintf (stderr, "GPU transpose: %Lg secs ==> %Lg billion elements/second\n",
           t_gpu, (N * N) / t_gpu * 1e-9 );
	fprintf (stdout, "GPU transpose: %Lg secs ==> %Lg billion elements/second\n",
	t_gpu, (N * N) / t_gpu * 1e-9 );

	double bw = (N * N * sizeof(dtype)) / (t_gpu * 1e9);
	fprintf (stdout, "Effective bandwidth: %.2lf GB/s\n", bw);

	CUDA_CHECK_ERROR (hipMemcpy (AT, d_odata, N * N * sizeof (dtype), 
	hipMemcpyDeviceToHost));

}

int 
main(int argc, char** argv)
{
  	/* variables */
	dtype *A, *ATgpu, *ATcpu;
	int err;
	int N;

  	struct stopwatch_t* timer = NULL;
  	long double t_cpu;

	N = -1;
	parseArg (argc, argv, &N);

  	/* input and output matrices on host */
  	/* output */
  	ATcpu = (dtype*) malloc (N * N * sizeof (dtype));
  	ATgpu = (dtype*) malloc (N * N * sizeof (dtype));

  	/* input */
  	A = (dtype*) malloc (N * N * sizeof (dtype));

	initArr (A, N * N);

	/* GPU transpose kernel */
	gpuTranspose (A, ATgpu, N);

  	/* Setup timers */
  	stopwatch_init ();
  	timer = stopwatch_create ();

	stopwatch_start (timer);
  	/* compute reference array */
	cpuTranspose (A, ATcpu, N);
  	t_cpu = stopwatch_stop (timer);
  	fprintf (stderr, "Time to execute CPU transpose kernel: %Lg secs\n",
		   t_cpu);
	fprintf (stdout, "Time to execute CPU transpose kernel: %Lg secs\n",
	t_cpu);


  	/* check correctness */
	err = cmpArr (ATgpu, ATcpu, N * N);
	if(err) {
		fprintf (stderr, "Transpose failed: %d\n", err);
		fprintf (stdout, "Transpose failed: %d\n", err);
	} else {
		fprintf (stderr, "Transpose successful\n");
		fprintf (stdout, "Transpose successful\n");
	}

	free (A);
	free (ATgpu);
	free (ATcpu);

  return 0;
}
