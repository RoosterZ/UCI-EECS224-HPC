#include "hip/hip_runtime.h"
#include <stdlib.h>
#include <stdio.h>

#include "cuda_utils.h"
#include "timer.c"

typedef float dtype;

#define BLOCK_DIM_Y 8
#define PATCH_DIM 32

// __global__ 
// void matTrans(dtype* AT, dtype* A, int N)  {
// 	/* Fill your code here */
// 	__shared__ dtype scratch[PATCH_DIM][PATCH_DIM+1];
// 	int x = blockIdx.x * PATCH_DIM + threadIdx.x;
// 	int y = blockIdx.y * PATCH_DIM + threadIdx.y;

// 	int i;
// 	// for (i = 0; i < PATCH_DIM; i += BLOCK_DIM_Y){
// 	// 	scratch[i + threadIdx.y][threadIdx.x] = A[(y+i) * N + x]; 
// 	// }

// 	for (i = 0; i < PATCH_DIM; i += BLOCK_DIM_Y) {
// 		scratch[i + threadIdx.y][threadIdx.x] = A[(y+i) * N + x]; 
// 	}

// 	//__syncthreads();

// 	x = PATCH_DIM * blockIdx.y + threadIdx.x;
// 	y = blockIdx.x * PATCH_DIM + threadIdx.y;

// 	__syncthreads();
 
// 	for (i = 0; i < PATCH_DIM; i += BLOCK_DIM_Y) {
// 		AT[(y+i) * N + x] = scratch[threadIdx.x][i + threadIdx.y];
// 	}
// }

__global__ 
void matTrans(dtype* AT, dtype* A, int N)  {
	/* Fill your code here */
	__shared__ dtype scratch[PATCH_DIM][PATCH_DIM+1];
	int x = blockIdx.x * PATCH_DIM + threadIdx.x;
	int y = N * (blockIdx.y * PATCH_DIM + threadIdx.y);
	int incy = BLOCK_DIM_Y * N;
	int i;

	for (i = 0; i < PATCH_DIM; i += BLOCK_DIM_Y, y += incy) {
		scratch[threadIdx.y + i][threadIdx.x] = A[y + x]; 
	}

	//__syncthreads();

	x = PATCH_DIM * blockIdx.y + threadIdx.x;
	y = N * (blockIdx.x * PATCH_DIM + threadIdx.y);

	__syncthreads();
 
	for (i = 0; i < PATCH_DIM; i += BLOCK_DIM_Y, y += incy) {
		AT[y + x] = scratch[threadIdx.x][threadIdx.y + i];
	}
}


void
parseArg (int argc, char** argv, int* N)
{
	if(argc == 2) {
		*N = atoi (argv[1]);
		assert (*N > 0);
	} else {
		fprintf (stderr, "usage: %s <N>\n", argv[0]);
		exit (EXIT_FAILURE);
	}
}

void
initArr (dtype* in, int N)
{
	int i;

	for(i = 0; i < N; i++) {
		in[i] = (dtype) rand () / RAND_MAX;
	}
}

void
cpuTranspose (dtype* A, dtype* AT, int N)
{
	int i, j;

	for(i = 0; i < N; i++) {
		for(j = 0; j < N; j++) {
			AT[j * N + i] = A[i * N + j];
		}
	}
}

int
cmpArr (dtype* a, dtype* b, int N)
{
	int cnt, i;

	cnt = 0;
	for(i = 0; i < N; i++) {
		if(abs(a[i] - b[i]) > 1e-6) cnt++;
	}

	return cnt;
}

void
gpuTranspose (dtype* A, dtype* AT, int N)
{
	dtype *d_idata, *d_odata;
	CUDA_CHECK_ERROR (hipMalloc (&d_idata, N * N * sizeof (dtype)));
	CUDA_CHECK_ERROR (hipMalloc (&d_odata, N * N * sizeof (dtype)));
	CUDA_CHECK_ERROR (hipMemcpy (d_idata, A, N * N * sizeof (dtype), 
	hipMemcpyHostToDevice));

	// int block_x, block_y, grid_x, grid_y;
	// getNumBlocksAndThreads(N, block_x, block_y, grid_x, grid_y);
	// dim3 gb(grid_x, grid_y, 1);
	// dim3 tb(block_x, block_y, 1);
	dim3 gb(N / PATCH_DIM, N / PATCH_DIM, 1);
	dim3 tb(PATCH_DIM, BLOCK_DIM_Y, 1);
	matTrans <<<gb, tb>>> (d_odata, d_idata, N);

	struct stopwatch_t* timer = NULL;
  	long double t_gpu;
	
  	/* Setup timers */
  	stopwatch_init ();
  	timer = stopwatch_create ();
  
	stopwatch_start (timer);
	  
	
	/* run your kernel here */
	matTrans <<<gb, tb>>> (d_odata, d_idata, N);

  	hipDeviceSynchronize ();
  	t_gpu = stopwatch_stop (timer);
  	fprintf (stderr, "GPU transpose: %Lg secs ==> %Lg billion elements/second\n",
           t_gpu, (N * N) / t_gpu * 1e-9 );
	fprintf (stdout, "GPU transpose: %Lg secs ==> %Lg billion elements/second\n",
	t_gpu, (N * N) / t_gpu * 1e-9 );

	double bw = (N * N * sizeof(dtype)) / (t_gpu * 1e9);
	fprintf (stdout, "Effective bandwidth: %.2lf GB/s\n", bw);

	CUDA_CHECK_ERROR (hipMemcpy (AT, d_odata, N * N * sizeof (dtype), 
	hipMemcpyDeviceToHost));

}

int 
main(int argc, char** argv)
{
  /* variables */
	dtype *A, *ATgpu, *ATcpu;
  int err;

	int N;

  struct stopwatch_t* timer = NULL;
  long double t_cpu;

	N = -1;
	parseArg (argc, argv, &N);

  /* input and output matrices on host */
  /* output */
  ATcpu = (dtype*) malloc (N * N * sizeof (dtype));
  ATgpu = (dtype*) malloc (N * N * sizeof (dtype));

  /* input */
  A = (dtype*) malloc (N * N * sizeof (dtype));

	initArr (A, N * N);

	/* GPU transpose kernel */
	gpuTranspose (A, ATgpu, N);

  /* Setup timers */
  stopwatch_init ();
  timer = stopwatch_create ();

	stopwatch_start (timer);
  /* compute reference array */
	cpuTranspose (A, ATcpu, N);
  t_cpu = stopwatch_stop (timer);
  fprintf (stderr, "Time to execute CPU transpose kernel: %Lg secs\n",
           t_cpu);

  /* check correctness */
	err = cmpArr (ATgpu, ATcpu, N * N);
	if(err) {
		fprintf (stderr, "Transpose failed: %d\n", err);
	} else {
		fprintf (stderr, "Transpose successful\n");
	}

	free (A);
	free (ATgpu);
	free (ATcpu);

  return 0;
}