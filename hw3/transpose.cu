#include "hip/hip_runtime.h"
#include <stdlib.h>
#include <stdio.h>

#include "cuda_utils.h"
#include "timer.c"

typedef float dtype;

#define BLOCK_DIM_Y 8
#define PATCH_DIM 32

__global__ 
void matTrans(dtype* AT, dtype* A, int N)  {
	/* Fill your code here */
	//const unsigned int scratch_dim = blockDim.x;
	
	__shared__ dtype scratch[PATCH_DIM][PATCH_DIM+1];
	int x = blockIdx.x * PATCH_DIM + threadIdx.x;
	int y = blockIdx.y * PATCH_DIM + threadIdx.y;

	 int i;
	// //int dim = gridDim.x * blockDim.x;
	for (i = 0; i < PATCH_DIM; i += BLOCK_DIM_Y){
		scratch[i + threadIdx.y][threadIdx.x] = A[(y+i) * N + x]; 
	}
	// for (i = 0; i < PATCH_DIM; i += BLOCK_DIM_Y){
	// 	AT[(y+i) * N + x] = A[(y+i) * N + x]; 
	// }	

	__syncthreads();

	x = PATCH_DIM * blockIdx.y + threadIdx.x;
	y = blockIdx.x * PATCH_DIM + threadIdx.y;
 
	for (i = 0; i < PATCH_DIM; i += BLOCK_DIM_Y){
		AT[(y+i) * N + x] = scratch[threadIdx.x][i + threadIdx.y];
	}
    

}


void
parseArg (int argc, char** argv, int* N)
{
	if(argc == 2) {
		*N = atoi (argv[1]);
		assert (*N > 0);
	} else {
		fprintf (stderr, "usage: %s <N>\n", argv[0]);
		exit (EXIT_FAILURE);
	}
}

void
initArr (dtype* in, int N)
{
	int i;

	for(i = 0; i < N; i++) {
		in[i] = (dtype) rand () / RAND_MAX;
	}
}

void
cpuTranspose (dtype* A, dtype* AT, int N)
{
	int i, j;

	for(i = 0; i < N; i++) {
		for(j = 0; j < N; j++) {
			AT[j * N + i] = A[i * N + j];
		}
	}
}

int
cmpArr (dtype* a, dtype* b, int N)
{
	int cnt, i;

	cnt = 0;
	for(i = 0; i < N; i++) {
		if(abs(a[i] - b[i]) > 1e-6) cnt++;
	}

	return cnt;
}

void
gpuTranspose (dtype* A, dtype* AT, int N)
{
	hipEvent_t startEvent, stopEvent;
	checkCuda( hipEventCreate(&startEvent) );
	checkCuda( hipEventCreate(&stopEvent) );
	float ms;

	dtype *d_idata, *d_odata;
	CUDA_CHECK_ERROR (hipMalloc (&d_idata, N * N * sizeof (dtype)));
	CUDA_CHECK_ERROR (hipMalloc (&d_odata, N * N * sizeof (dtype)));
	CUDA_CHECK_ERROR (hipMemcpy (d_idata, A, N * N * sizeof (dtype), 
	hipMemcpyHostToDevice));

	// int block_x, block_y, grid_x, grid_y;
	// getNumBlocksAndThreads(N, block_x, block_y, grid_x, grid_y);
	// dim3 gb(grid_x, grid_y, 1);
	// dim3 tb(block_x, block_y, 1);
	dim3 gb(32, 32, 1);
	dim3 tb(32, 8, 1);
	matTrans <<<gb, tb>>> (d_odata, d_idata, N);

	struct stopwatch_t* timer = NULL;
  	long double t_gpu;
	
  	/* Setup timers */
  	stopwatch_init ();
  	timer = stopwatch_create ();
  
	stopwatch_start (timer);
	CUDA_CHECK_ERROR( hipEventRecord(startEvent, 0));  
	matTrans <<<gb, tb>>> (d_odata, d_idata, N);
	/* run your kernel here */
	CUDA_CHECK_ERROR( hipEventRecord(stopEvent, 0) );
	CUDA_CHECK_ERROR( hipEventSynchronize(stopEvent) );
	CUDA_CHECK_ERROR( hipEventElapsedTime(&ms, startEvent, stopEvent) );

  	hipDeviceSynchronize ();
  	t_gpu = stopwatch_stop (timer);
  	fprintf (stderr, "GPU transpose: %Lg secs ==> %Lg billion elements/second\n",
           t_gpu, (N * N) / t_gpu * 1e-9 );
	fprintf (stdout, "GPU transpose: %Lg secs ==> %Lg billion elements/second\n",
	t_gpu, (N * N) / t_gpu * 1e-9 );


	// double bw = (N * N * sizeof(dtype)) / (t_gpu * 1e9);
	// fprintf (stdout, "Effective bandwidth: %.2lf GB/s\n", bw);
	double bw = (N * N * sizeof(dtype)) / (ms * 1e6);
	fprintf (stdout, "Effective bandwidth: %.2lf GB/s\n", bw);

	CUDA_CHECK_ERROR (hipMemcpy (AT, d_odata, N * N * sizeof (dtype), 
	hipMemcpyDeviceToHost));

}

int 
main(int argc, char** argv)
{
  /* variables */
	dtype *A, *ATgpu, *ATcpu;
  int err;

	int N;

  struct stopwatch_t* timer = NULL;
  long double t_cpu;

	N = -1;
	parseArg (argc, argv, &N);

  /* input and output matrices on host */
  /* output */
  ATcpu = (dtype*) malloc (N * N * sizeof (dtype));
  ATgpu = (dtype*) malloc (N * N * sizeof (dtype));

  /* input */
  A = (dtype*) malloc (N * N * sizeof (dtype));

	initArr (A, N * N);

	/* GPU transpose kernel */
	gpuTranspose (A, ATgpu, N);

  /* Setup timers */
  stopwatch_init ();
  timer = stopwatch_create ();

	stopwatch_start (timer);
  /* compute reference array */
	cpuTranspose (A, ATcpu, N);
  t_cpu = stopwatch_stop (timer);
  fprintf (stderr, "Time to execute CPU transpose kernel: %Lg secs\n",
           t_cpu);

  /* check correctness */
	err = cmpArr (ATgpu, ATcpu, N * N);
	if(err) {
		fprintf (stderr, "Transpose failed: %d\n", err);
	} else {
		fprintf (stderr, "Transpose successful\n");
	}

	free (A);
	free (ATgpu);
	free (ATcpu);

  return 0;
}